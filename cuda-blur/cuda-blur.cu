#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <string.h>

using namespace cv;

__global__
void saxpy(int n, float a, float *x, float *y){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a*x[i] + y[i];
}

int main(int argc, char** argv){
    Mat image, result, *d_image, *d_result;
    int kernel;
    int height, width;

    if (argc != 3){
        printf("Use: ./blurSecuencial.out <img_path> <num_kernel>\n");
        return -1;
    }

    image = imread( argv[1], 1 );

    if ( !image.data ){
        printf("Image couldn't be read. \n");
        return -1;
    }

    kernel = atoi(argv[2]);

    height = image.size().height;
    width = image.size().width;

    // Create result image with same dimensions
    result = Mat(height, width, CV_8UC3);

    // CUDA
    hipMalloc(&d_image, sizeof(image));
    hipMalloc(&d_result, sizeof(image));

    hipMemcpy(d_image, &image, sizeof(image), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, sizeof(image), hipMemcpyHostToDevice);

    d_result = d_image;

    hipMemcpy(&result, d_result, sizeof(image), hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_result);

    ///////////////////////////////////////////

    int N = 1<<20;
    float *x, *y, *d_x, *d_y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    char file_name[12] = "blurred.jpg";
    imwrite(file_name, result);
}
